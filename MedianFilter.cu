#include "hip/hip_runtime.h"
#include "MedianFilter.cuh"

__device__ void csort(uint8_t ls[], int l, int r);

//
// GPU implementation
//
__global__ void MedianFilterGPUKernel(uint8_t* imageSrc, uint8_t* imageDst, int channels, int32_t uKernelRadius, uint32_t uKernelPixelCount, uint32_t uMedianIdx)
{
	int row = blockIdx.x;
	int col = blockIdx.y;
	int rows = gridDim.x;
	int cols = gridDim.y;

	//if (row > 0 && col > 0 && row < rows && col < cols)
	//{

	auto pPixelValues = new uint8_t[uKernelPixelCount];

	// get all pixel values
	int32_t i = 0;
	for (int32_t kernel_row = -uKernelRadius; kernel_row <= uKernelRadius; ++kernel_row)
		for (int32_t kernel_col = -uKernelRadius; kernel_col <= uKernelRadius; ++kernel_col)
			pPixelValues[++i] = imageSrc[(row + kernel_row) * cols + (col + kernel_col)];

	csort(pPixelValues, 0, uKernelPixelCount);
	imageDst[row * cols + col] = pPixelValues[uMedianIdx];
	//}
}


namespace filter
{
	void MedianFilterGPU(std::vector<uint8_t> const& imageSrc, std::vector<uint8_t>& imageDst, int rows, int cols, int channels, uint32_t uKernelSize)
	{
		int32_t uKernelRadius = uKernelSize / 2;
		uint32_t uKernelPixelCount = uKernelSize * uKernelSize;
		uint32_t uMedianIdx = (uKernelPixelCount + 1) / 2;

		uint8_t* dev_imageSrc;
		uint8_t* dev_imageDst;

		// allocate the memory on the GPU
		hipMalloc((void**)&dev_imageSrc, imageSrc.size() * sizeof(uint8_t));
		hipMalloc((void**)&dev_imageDst, imageDst.size() * sizeof(uint8_t));

		// fill the arrays �imageSrc� and �imageDst� on the GPU
		hipMemcpy(dev_imageSrc, imageSrc.data(), imageSrc.size() * sizeof(uint8_t), hipMemcpyHostToDevice);
		hipMemcpy(dev_imageDst, imageDst.data(), imageDst.size() * sizeof(uint8_t), hipMemcpyHostToDevice);

		// run filter
		dim3 grid(rows, cols);
		MedianFilterGPUKernel << < grid, 1 >> > (dev_imageSrc, dev_imageDst, channels, uKernelRadius, uKernelPixelCount, uMedianIdx);

		// copy data from GPU
		hipMemcpy(imageDst.data(), dev_imageDst, imageDst.size() * sizeof(uint8_t), hipMemcpyDeviceToHost);

		// cleanup
		hipFree(dev_imageSrc);
		hipFree(dev_imageDst);
	}
}


#define swap(A,B) { float temp = A; A = B; B = temp;}

//
// Quick sort
// thanks to https://github.com/khaman1/GPU-QuickSort-Algorithm
//
__device__ void csort(uint8_t ls[], int l, int r) {
	int i, j, k, p, q;
	float v;
	if (r <= l)
		return;
	v = ls[r];
	i = l - 1;
	j = r;
	p = l - 1;
	q = r;
	for (;;) {
		while (ls[++i] < v);
		while (v < ls[--j])
			if (j == l)
				break;
		if (i >= j)
			break;
		swap(ls[i], ls[j]);
		if (ls[i] == v) {
			p++;
			swap(ls[p], ls[i]);
		}
		if (v == ls[j]) {
			q--;
			swap(ls[q], ls[j]);
		}
	}
	swap(ls[i], ls[r]);
	j = i - 1;
	i++;
	for (k = l; k < p; k++, j--)
		swap(ls[k], ls[j]);
	for (k = r - 1; k > q; k--, i++)
		swap(ls[k], ls[i]);

	csort(ls, l, j);
	csort(ls, i, r);
}

